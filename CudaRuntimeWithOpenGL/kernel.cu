#include "hip/hip_runtime.h"
﻿// Include CUDA headers first
#include "hip/hip_runtime.h"
#include ""
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

// Then include OpenGL headers
#include <glad/glad.h>
#include <GLFW/glfw3.h>

//classic headers
#include <stdio.h>
#include <cstdlib>
#include <ctime>  
#include <cmath>
#include <iostream>

#define BoidCount 10
#define CELL_ARRAY_SIZE 25

#define WIDTH 500
#define HEIGHT 500

//boid logic parameters
#define PROTECTED_RANGE 20
#define AVOIDFACTOR 0.5

uint2 gridList[BoidCount];

struct vec2 {
    float x;
    float y;
};
class Cell {
public:
    int Id;
    vec2 position;

    Cell() : Id(0), position({ 0.0f, 0.0f }) {}
    Cell(int id, vec2 pos) : Id(id), position(pos) {}
};
class Boid {
public:
    int Id;
    vec2 position;
    float2 velocity;

    Boid() : Id(0), position({ 0.0f, 0.0f }), velocity({ 10,10 }) {}
    Boid(int id, vec2 pos) : Id(id), position(pos), velocity({ 10,10 }) {}
};
struct CompareX {
    __host__ __device__
        bool operator()(const uint2& a, const uint2& b) const {
        return a.x < b.x;
    }
};

void initializeBoids(int width, int height, Boid(&boidArray)[BoidCount], int size);
void SortGridList(uint2* d_gridList);
void initializeCells(int width, int height, Cell(&cellArray)[CELL_ARRAY_SIZE], int cellArraySize);
void CreateLookUpTable(uint2* d_gridList, Boid* d_boidArray, Cell* d_cellArray, int* d_lookUpTable);
void CalculateBoidLogic(uint2* d_gridList, Boid* d_boidArray, Cell* d_cellArray, int* d_lookUpTable, float deltaTime);
void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

void checkCudaError(hipError_t cudaStatus, const char* errorMessage) {
    if (cudaStatus != hipSuccess) {
        std::cerr << errorMessage << " failed: " << hipGetErrorString(cudaStatus) << "\n";
        exit(EXIT_FAILURE);
    }
}

__global__ void hashBoids(Boid* boidArray, uint2* gridList, Cell* cellArray, int boidCount, int cellArraySize, int width, int height) {
    int boidIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (boidIndex < boidCount) {
        // Calculate the index of the cell for the current boid based on its position
        int cellIndex = static_cast<int>((boidArray[boidIndex].position.y / height) * sqrt((float)cellArraySize)) * sqrt((float)cellArraySize)
            + static_cast<int>((boidArray[boidIndex].position.x / width) * sqrt((float)cellArraySize));
        gridList[boidIndex] = make_uint2(cellIndex, boidIndex);

        //checking if cell is a corner or a side
        // temporarry 
        width = 5;
        height = 5;

        int xOffset = cellIndex % width;
        int yOffset = static_cast<int>((cellIndex - xOffset) / height);

        //adding neighbors 
        int stride = boidCount;
        //left cell
        if (xOffset != 0)
            gridList[boidIndex + stride] = make_uint2(cellIndex - 1, boidIndex);

        //right
        if (xOffset != width - 1)
            gridList[boidIndex + stride * 2] = make_uint2(cellIndex + 1, boidIndex);

        //top cell
        if (yOffset != 0)
            gridList[boidIndex + stride * 3] = make_uint2(cellIndex - width, boidIndex);


        //bottom cell
        if (yOffset < height - 1)
            gridList[boidIndex + stride * 4] = make_uint2(cellIndex + width, boidIndex);

        //right-top cell
        if (yOffset != 0 && xOffset != width - 1)
            gridList[boidIndex + stride * 5] = make_uint2(cellIndex - width + 1, boidIndex);

        //right-bot cell
        if (yOffset < height - 1 && xOffset != width - 1)
            gridList[boidIndex + stride * 6] = make_uint2(cellIndex + width + 1, boidIndex);

        //left-top cell
        if (xOffset != 0 && yOffset != 0)
            gridList[boidIndex + stride * 7] = make_uint2(cellIndex - width - 1, boidIndex);

        //left-bot cell
        if (xOffset != 0 && yOffset < height - 1)
            gridList[boidIndex + stride * 8] = make_uint2(cellIndex + width - 1, boidIndex);
    }
}
__global__ void makeLookupTable(uint2* gridList, Boid* boidArray, Cell* cellArray, int* lookUpTable)
{
    int gridListIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (gridList[gridListIndex].x == 404)
        return;

    if (gridListIndex < BoidCount)
    {
        if (gridListIndex == 0)
        {
            lookUpTable[gridList[gridListIndex].x] = gridListIndex;
            return;
        }

        if (gridList[gridListIndex].x > gridList[gridListIndex - 1].x)
        {
            lookUpTable[gridList[gridListIndex].x] = gridListIndex;
            return;
        }
    }
    else
    {
        if (gridListIndex == BoidCount)
        {
            lookUpTable[gridList[gridListIndex].x + CELL_ARRAY_SIZE] = gridListIndex;
        }
        if (gridList[gridListIndex].x > gridList[gridListIndex - 1].x)
        {
            lookUpTable[gridList[gridListIndex].x + CELL_ARRAY_SIZE] = gridListIndex;
            return;
        }
    }

}

__device__ float2 calculateSeparation(int* localBoidIDs, int* neighboringBoidIds, int boidIndex, Boid* boids, int localBoidIDsSize, int neighboringBoidIdsSize) {
    int closeDx = 0, closeDy = 0;
    //retrive the boid from boidIds 
    Boid currentBoid = boids[localBoidIDs[boidIndex]];
    //loop through local boids
    for (int i = 0; i < localBoidIDsSize; i++)
    {
        if (i != boidIndex)
        {
            float distX = currentBoid.position.x - boids[localBoidIDs[i]].position.x;
            float distY = currentBoid.position.y - boids[localBoidIDs[i]].position.y;
            //add only if inside protected range
            if (sqrt(distX * distX + distY * distY) < PROTECTED_RANGE)
            {
                closeDx += distX;
                closeDy += distY;
            }

        }
    }
    //loop through neighboring boids 
    for (int i = 0; i < neighboringBoidIdsSize; i++)
    {
        float distX = currentBoid.position.x - boids[neighboringBoidIds[i]].position.x;
        float distY = currentBoid.position.y - boids[neighboringBoidIds[i]].position.y;
        //add only if inside protected range
        if (sqrt(distX * distX + distY * distY) < PROTECTED_RANGE)
        {
            closeDx += distX;
            closeDy += distY;
        }
    }

    return make_float2(closeDx, closeDy);
}
__global__ void calculateBoidLogic(uint2* gridList, Boid* boidArray, Cell* cellArray, int* lookUpTable, float deltaTime)
{
    //int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

    //hard coded estimated values. might need to adjusted acordings with the Boid count 
    __shared__ int localBoidIds[50];
    __shared__ int neighboringBoidIds[50 * 8];

    //for testing delete later!!!!!
    const int sentinelValue = -1;
    for (int i = 0; i < 50; ++i) {
        localBoidIds[i] = sentinelValue;
    }

    //check if there exists boids in the cell
    if (lookUpTable[blockIdx.x] == -1)
        return;

    //determines the size of boidIndex and neighboringBoidIds array
    int localBoidIndexBoids = 0;
    int neighboringBoids = 0;

    //calculate local boids
    if (threadIdx.x == 33)
    {
        int lookupIndex = lookUpTable[blockIdx.x];
        uint2 cellBoid = gridList[lookupIndex];

        //debug vars
        int xValue = gridList[lookupIndex].x;
        int yValue = gridList[lookupIndex].y;

        int oldCellId = cellBoid.x;


        while (oldCellId == cellBoid.x)
        {
            oldCellId = cellBoid.x;
            localBoidIds[localBoidIndexBoids] = cellBoid.y;

            //printf("Block ID: %d, Thread ID: %d, localBoidIndexBoids: %d, cellBoid.x: %f, cellBoid.y: %f, xValue: %d, yValue: %d, lookupIndex: %d\n",
            //    blockIdx.x, threadIdx.x, localBoidIndexBoids,
            //    cellBoid.x, cellBoid.y, xValue, yValue,
            //    lookupIndex);

            localBoidIndexBoids++;
            lookupIndex++;
            cellBoid = gridList[lookupIndex];
            
            xValue = gridList[lookupIndex].x;
            yValue = gridList[lookupIndex].y;
        }
    }
    //calculate neighbor boids
    if (threadIdx.x == 34)
    {
        int lookupIndex = lookUpTable[blockIdx.x + CELL_ARRAY_SIZE];
        uint2 cellBoid = gridList[lookupIndex];
        int oldCellId = cellBoid.x;

        while (oldCellId == cellBoid.x)
        {
            oldCellId = cellBoid.x;
            neighboringBoidIds[neighboringBoids] = cellBoid.y;
            neighboringBoids++;
            lookupIndex++;
            cellBoid = gridList[lookupIndex];
        }
    }
    __syncthreads();
    //__________________________________________________________________PROBLEM IS THAT ONLY THREAD ID 0 CONTINUES ALSO THERE IS A MEMORRY OVERFLOW IN LOOKUP TABLE  ____________________________________
    int boidIndex = threadIdx.x;
    if (localBoidIds[boidIndex] == -1)
    {
        //printf("Block ID: %d, Thread ID: %d. Failed\n", blockIdx.x, threadIdx.x);
        return;
    }
    //printf("Block ID: %d, Thread ID: %d. Passed\n", blockIdx.x, threadIdx.x);
        
    int currentBoidId = localBoidIds[boidIndex];
    Boid& currentBoid = boidArray[currentBoidId];

    // Fish logic: Update velocity based on separation, alignment, and cohesion rules
    float2 separation = calculateSeparation(localBoidIds, neighboringBoidIds, boidIndex, boidArray, localBoidIndexBoids, neighboringBoids);

    // Weighting factors for each rule
    float alignmentWeight = 1.0;
    float cohesionWeight = 1.0;

    // Update velocity based on the rules
    //boidArray[boidIndex].velocity += separationWeight * separation +
    //    alignmentWeight * alignment +
    //    cohesionWeight * cohesion;

    float edgeAvoidanceFactor = 1;
    float safeDistance = 10.0;      

    //edge detect
    if (currentBoid.position.x < safeDistance)
        separation.x += edgeAvoidanceFactor;
    else if (currentBoid.position.x > WIDTH - safeDistance)
        separation.x -= edgeAvoidanceFactor;

    if (currentBoid.position.y < safeDistance)
        separation.y += edgeAvoidanceFactor;
    else if (currentBoid.position.y > HEIGHT - safeDistance)
        separation.y -= edgeAvoidanceFactor;

    currentBoid.velocity.x += separation.x * AVOIDFACTOR;
    currentBoid.velocity.y += separation.y * AVOIDFACTOR;

    //speed limit
    if (currentBoid.velocity.x > 10)
        currentBoid.velocity.x = 10;
    if (currentBoid.velocity.y > 10)
        currentBoid.velocity.y = 10;

    if (currentBoid.velocity.x < -10)
        currentBoid.velocity.x = -10;
    if (currentBoid.velocity.y < -10)
        currentBoid.velocity.y = -10;

    //debug code
    //boidArray[localBoidIds[boidIndex]].velocity.x += 1 * AVOIDFACTOR;
    //boidArray[localBoidIds[boidIndex]].velocity.y += 1 * AVOIDFACTOR;

    //printf("Block ID: %d, Thread ID: %d, Boid ID: %d, Position: (%f, %f), Velocity: (%f, %f)\n",
    //    blockIdx.x, threadIdx.x, currentBoidId,
    //    currentBoid.position.x, currentBoid.position.y,
    //    currentBoid.velocity.x, currentBoid.velocity.y);

    currentBoid.position.x += currentBoid.velocity.x * deltaTime;
    currentBoid.position.y += currentBoid.velocity.y * deltaTime;

    __syncthreads();
}
__global__ void setUint2Values(uint2* array, int value, int count) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < count) {
        array[index].x = value;
        array[index].y = value;
    }
}

//shader programs
const char* vertexShaderSource = R"(
    #version 330 core
    layout (location = 0) in vec2 aPos; // Use vec2 for 2D positions
    void main() {
        gl_Position = vec4(aPos.x, aPos.y, 0.0, 1.0);
    }
)";

const char* fragmentShaderSource = R"(
    #version 330 core
    out vec4 FragColor;
    void main() {
        FragColor = vec4(1.0f, 0.5f, 0.2f, 1.0f);
    }
)";


int main()
{
    Boid boidArray[BoidCount];

    // Call the function to initialize Boids
    initializeBoids(WIDTH, HEIGHT, boidArray, BoidCount);

    // Access and use the initialized Boids
    //std::cout << "Boids array____________________________________________________________________________________" << std::endl;
    //for (const auto& boid : boidArray) {
    //    std::cout << "Boid Id: " << boid.Id << ", Position: (" << boid.position.x << ", " << boid.position.y << ")\n";
    //}

    Cell cellArray[CELL_ARRAY_SIZE];

    // Call the function to initialize Cells
    initializeCells(WIDTH, HEIGHT, cellArray, CELL_ARRAY_SIZE);

    // Access and use the initialized Cells
    //std::cout << "Cell array____________________________________________________________________________________" << std::endl;
    //for (const auto& cell : cellArray) {
    //    std::cout << "Cell Id: " << cell.Id << ", Position: (" << cell.position.x << ", " << cell.position.y << ")\n";
    //}

    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    hipSetDevice(0);

    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "LearnOpenGL", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }

    glViewport(0, 0, WIDTH, HEIGHT);

    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    unsigned int VAO, VBO;
    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);

    glGenBuffers(1, &VBO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, BoidCount * sizeof(vec2), nullptr, GL_DYNAMIC_DRAW);

    unsigned int vertexShader;
    vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);

    unsigned int fragmentShader;
    fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);

    int  success;
    char infoLog[512];
    glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
    if (!success)
    {
        glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
        std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
    }

    glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
    if (!success)
    {
        glGetShaderInfoLog(fragmentShader, 512, NULL, infoLog);
        std::cout << "ERROR::SHADER::FRAG::COMPILATION_FAILED\n" << infoLog << std::endl;
    }

    unsigned int shaderProgram;
    shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);

    glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
    if (!success) {
        glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
        std::cout << "shader program compliation failed\n" << infoLog << std::endl;
    }
    
    glUseProgram(shaderProgram);
    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);

    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(vec2), (void*)0);
    glEnableVertexAttribArray(0);

    Boid* d_boidArray;
    uint2* d_gridList;
    Cell* d_cellArray;
    int* d_lookUpTable;
    std::vector<vec2> renderArray(BoidCount);

    hipMalloc((void**)&d_boidArray, BoidCount * sizeof(Boid));
    hipMalloc((void**)&d_gridList, BoidCount * 9 * sizeof(uint2));//since we will add boids to multiple cells we can have at max 4 * BoidCount lenght gridList!
    hipMalloc((void**)&d_cellArray, CELL_ARRAY_SIZE * sizeof(Cell));

    // Copy boidArray and cellArray to device memory
    hipMemcpy(d_boidArray, boidArray, BoidCount * sizeof(Boid), hipMemcpyHostToDevice);
    hipMemcpy(d_cellArray, cellArray, CELL_ARRAY_SIZE * sizeof(Cell), hipMemcpyHostToDevice);

    //main simulation loop
    double lastFrameTime = glfwGetTime();
    while (!glfwWindowShouldClose(window))
    {
        // Calculate delta time
        double currentFrameTime = glfwGetTime();
        double deltaTime = currentFrameTime - lastFrameTime;
        lastFrameTime = currentFrameTime;

        setUint2Values << <1, BoidCount * 9 >> > (d_gridList, 404, BoidCount * 9);
        hipDeviceSynchronize();

        hashBoids << <1, BoidCount >> > (d_boidArray, d_gridList, d_cellArray, BoidCount, CELL_ARRAY_SIZE, WIDTH, HEIGHT);
        hipDeviceSynchronize();
        //sort grid list
        SortGridList(d_gridList);
        //std::cout << "sorted array____________________________________________________________________________________" << std::endl;
        //uint2* hostGridList = new uint2[BoidCount * 9];
        //hipMemcpy(hostGridList, d_gridList, BoidCount * sizeof(uint2) * 9, hipMemcpyDeviceToHost);
        //int k = 0;
        //for (int i = 0; i < 9; i++)
        //{
        //    for (int j = 0; j < BoidCount; j++)
        //    {
        //        std::cout << "cell Id: " << hostGridList[k].x << " boid Id: " << hostGridList[k].y << std::endl;
        //        k++;
        //    }
        //}
        hipDeviceSynchronize();

        //init lookup table array
        
        checkCudaError(hipMalloc((void**)&d_lookUpTable, CELL_ARRAY_SIZE * 2 * sizeof(int)), "mallocFailed");
        hipMemset(d_lookUpTable, -1, CELL_ARRAY_SIZE * 2 * sizeof(int));

        //create look up table
        CreateLookUpTable(d_gridList, d_boidArray, d_cellArray, d_lookUpTable);
        hipDeviceSynchronize();
        //std::cout << "LookUp table____________________________________________________________________________________" << std::endl;
        //int* h_lookUptable = new int[CELL_ARRAY_SIZE * 2];
        //hipMemcpy(h_lookUptable, d_lookUpTable, CELL_ARRAY_SIZE * 2 * sizeof(int), hipMemcpyDeviceToHost);
        //for (int i = 0; i < CELL_ARRAY_SIZE * 2; ++i)
        //    std::cout << "Cell index: " << i % CELL_ARRAY_SIZE << " starts at index: " << h_lookUptable[i] << std::endl;


        CalculateBoidLogic(d_gridList, d_boidArray, d_cellArray, d_lookUpTable, deltaTime);
        hipDeviceSynchronize();
        hipMemcpy(boidArray, d_boidArray, BoidCount * sizeof(Boid), hipMemcpyDeviceToHost);


        // Extract only positions for rendering
        for (int i = 0; i < BoidCount; ++i)
        {
            float normalized_x = (2.0f * boidArray[i].position.x / WIDTH) - 1.0f;
            float normalized_y = 1.0f - (2.0f * boidArray[i].position.y / HEIGHT);

            renderArray[i].x = normalized_x;
            renderArray[i].y = normalized_y;
         }


        // Update the VBO with the new boid positions
        glBindBuffer(GL_ARRAY_BUFFER, VBO);
        glBufferSubData(GL_ARRAY_BUFFER, 0, BoidCount * sizeof(vec2), renderArray.data());
        //renderArray.clear();

        // OpenGL rendering
        glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // Use your shader program and VAO here
        glUseProgram(shaderProgram);
        glBindVertexArray(VAO);
        glPointSize(5.0f); // Set point size to make boids larger
        glDrawArrays(GL_POINTS, 0, BoidCount); // Assuming you want to draw points for each boid

        glfwPollEvents();
        glfwSwapBuffers(window);

        //std::cout << std::endl;
        //std::cout << "NEXT FRAME ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++" << std::endl;
        //std::cout << std::endl;
    }
    // Free allocated memory on the device
    hipFree(d_boidArray);
    hipFree(d_gridList);
    hipFree(d_cellArray);
    hipFree(d_lookUpTable);
    delete[] boidArray;

    glfwTerminate();
    return 0;
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}
void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}
void CalculateBoidLogic(uint2* d_gridList, Boid* d_boidArray, Cell* d_cellArray, int* d_lookUpTable, float deltaTime)
{
    calculateBoidLogic << < CELL_ARRAY_SIZE, 100 >> > (d_gridList, d_boidArray, d_cellArray, d_lookUpTable, deltaTime);
}
void CreateLookUpTable(uint2* d_gridList, Boid* d_boidArray, Cell* d_cellArray, int* d_lookUpTable)
{
    makeLookupTable << <1, BoidCount * 9 >> > (d_gridList, d_boidArray, d_cellArray, d_lookUpTable);
}
void SortGridList(uint2* d_gridList)
{
    thrust::device_ptr<uint2> dev_ptr_gridList(d_gridList);

    // Sort based on x values
    thrust::sort(dev_ptr_gridList, dev_ptr_gridList + BoidCount, CompareX());
    thrust::sort(dev_ptr_gridList + BoidCount, dev_ptr_gridList + BoidCount * 9, CompareX());

    // Copy the sorted data back to the host if needed
    hipMemcpy(d_gridList, thrust::raw_pointer_cast(dev_ptr_gridList), BoidCount * 9 * sizeof(uint2), hipMemcpyDeviceToDevice);
}
void initializeBoids(int width, int height, Boid(&boidArray)[BoidCount], int size) {
    // Seed the random number generator
    std::srand(static_cast<unsigned int>(std::time(0)));

    for (int i = 0; i < size; ++i) {
        // Assign random positions within the specified width and height
        float randomX = static_cast<float>(std::rand() % width);
        float randomY = static_cast<float>(std::rand() % height);

        // Assign sequential Ids
        int id = i;

        // Initialize the Boid with the generated values
        boidArray[i] = Boid(id, { randomX, randomY});
    }
}
void initializeCells(int width, int height, Cell(&cellArray)[CELL_ARRAY_SIZE], int cellArraySize) {
    // Calculate the size of each cell based on the grid and number of cells
    float cellWidth = static_cast<float>(width) / static_cast<float>(sqrt(cellArraySize));
    float cellHeight = static_cast<float>(height) / static_cast<float>(sqrt(cellArraySize));
    std::cout << cellWidth << " " << cellHeight << std::endl;
    int k = 0;
    // Loop through the cell array and initialize each cell
    for (int i = 0; i < HEIGHT / cellHeight; i++) {
        for (int j = 0; j < WIDTH / cellWidth; j++)
        {
            float cellX = static_cast<float>(j * cellWidth + cellWidth / 2);
            float cellY = static_cast<float>(i * cellWidth + cellWidth / 2);
            cellArray[k] = Cell{ k, {cellX, cellY} };
            k++;
        }
    }
}

